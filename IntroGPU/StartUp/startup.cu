
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;


__global__ void kernel_call()
{
  
}


int main(){

    float *host_in;
    float *dev_in;

    size_t N = 1<<2;

    hipEvent_t st1, et1, st2, et2;
    hipEventCreate(&st1);
    hipEventCreate(&et1);
    hipEventCreate(&st2);
    hipEventCreate(&et2);
    
    float ms1, ms2;
    
    //create buffer on host
    host_in = (float*) malloc(N * sizeof(float));
    
    //create buffer on device
    hipError_t err = hipMalloc(&dev_in, N*sizeof(float));
    if (err != hipSuccess){
       cout<<"Dev Memory not allocated"<<endl;
       exit(-1);
     }

    //record time at start
    hipEventRecord(st1);
    hipMemcpy(dev_in, host_in, sizeof(float)*N, hipMemcpyHostToDevice);

    //no sync required here because memcpy is synchronized
    hipEventRecord(et1);    

    //record time at start
    hipEventRecord(st2);
    
    kernel_call<<<4, 1024>>>();

    //wait until kernel is done start timing
    hipDeviceSynchronize();       
    hipEventRecord(et2);


    hipEventElapsedTime(&ms1, st1, et1);
    hipEventElapsedTime(&ms2, st2, et2);


    cout<<"MemCpy: "<<N<<" floats:\t"<<ms1<<"ms"<<endl;
    cout<<"Kernel:\t\t\t"<<ms2<<"ms"<<endl;    
    
    free(host_in);
    hipFree(dev_in);

  return 0;
}
