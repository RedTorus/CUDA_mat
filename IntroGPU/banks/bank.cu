
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>
#include <stdlib.h>
#include <stdio.h>

using namespace std;


__global__ void kernel_call(int N, float *in, float* out)
{
   __shared__ float share_buf[64*64];

   //DO NOT CHANGE ANY CODE ABOVE THIS COMMENT
   
   int id = threadIdx.x; //get my id;

   //read 2 rows at a time, write 2 columns at a time
   for (int i = 0; i != 64*64/blockDim.x; ++i)
      share_buf[i*2 + (id % 64)*64 + (id / 64)] = in[id + blockDim.x*i]; 
 
   __syncthreads();  //wait till everyone is done

   //copy everything to main memory
   for (int i = 0; i != 64*64/blockDim.x; ++i)
      out[id + blockDim.x * i] = share_buf[id + blockDim.x*i]; 
   }


int main(){

    float *host_in, *host_out;
    float *dev_in, *dev_out;

    size_t N = 64;
		
    //create buffer on host	
    host_in = (float*) malloc(N * N * sizeof(float));
    host_out = (float*) malloc(N * N * sizeof(float));

    //creates a matrix stored in row major order
    for (int i = 0; i != N; ++i)
        for (int j = 0; j != N; ++j)
     	  host_in[i*N + j] = i*N + j;   


    //create buffer on device
    hipError_t err = hipMalloc(&dev_in, N*N*sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out, N*N*sizeof(float));
    if (err != hipSuccess){
       cout<<"Dev Memory not allocated"<<endl;
       exit(-1);
    }
     
    hipMemcpy(dev_in, host_in, N * N * sizeof(float), hipMemcpyHostToDevice);

    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
     
    hipEventRecord(st2);
    kernel_call<<<1, 128>>>(N, dev_in, dev_out);
    hipEventRecord(et2);
        
    //host waits until et2 has occured     
    hipEventSynchronize(et2);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st2, et2);

    cout<<"Kernel time: "<<milliseconds<<"ms"<<endl;

    //copy data out
    hipMemcpy(host_out, dev_out, N * N * sizeof(float), hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i != N; ++i)
       for (int j = 0; j != N; ++j)
          correct &= (host_out[i*N+j] == host_in[j*N+i]);
    cout<<(correct ? "Yes" : "No")<<endl;	 
   
    hipEventDestroy(st2);
    hipEventDestroy(et2);

    free(host_in);
    free(host_out);
    hipFree(dev_in);
    hipFree(dev_out);

  return 0;
}
