
#include <hip/hip_runtime.h>
#include <iostream>
#include <omp.h>
#include <stdlib.h>
#include <stdio.h>

using namespace std;


__global__ void kernel_call(int N, float *in, float* out)
{
   __shared__ float share_buf[64*64];

   //DO NOT CHANGE ANY CODE ABOVE THIS COMMENT
   
   int tilex = blockIdx.x; // tile column
   int tiley = blockIdx.y; // tile row
   int id = threadIdx.x;

   //64*64/blocdim.x = number of elements to be processed per Thread

   //read 2 rows at a time, write 2 columns at a time
   // for (int i = 0; i != 64*64/blockDim.x; ++i)
   //    share_buf[i*2 + (id % 64)*64 + (id / 64)] = in[id + blockDim.x*i]; 
   in = in + tilex*64*64 + N*64*tiley;
   out = out + tiley*64*64 + N*64*tilex;

   int offset = (id/32)*16;
   int fac= 64; //blockDim.x/4; //32
   int col;
   int row;
   for (int i = 0; i != 64*64/blockDim.x; ++i){
      //share_buf[offset + (id % 32) + i * fac] = in[id + blockDim.x * i];
      col = (i%2)*32;
      row= i/2;
      share_buf[row + offset + fac*( (id%32) +col)] = in[id%32 + col + fac*(row +offset)];
      //share_buf[offset + row + (id % 32) + i * fac] = in[id + blockDim.x * i];
   }   
 
   __syncthreads();  //wait till everyone is done

   //copy everything to main memory
   for (int i = 0; i != 64*64/blockDim.x; ++i)
      out[id + blockDim.x * i] = share_buf[id + blockDim.x*i]; 
   }


int main(){

    float *host_in, *host_out;
    float *dev_in, *dev_out;

    size_t N = 128;
		
    //create buffer on host	
    host_in = (float*) malloc(N * N * sizeof(float));
    host_out = (float*) malloc(N * N * sizeof(float));

    //creates a matrix stored in row major order
    for (int i = 0; i != N; ++i)
        for (int j = 0; j != N; ++j)
     	  host_in[i*N + j] = i*N + j;   


    //create buffer on device
    hipError_t err = hipMalloc(&dev_in, N*N*sizeof(float));
    if (err != hipSuccess){
      cout<<"Dev Memory not allocated"<<endl;
      exit(-1);
    }

    err = hipMalloc(&dev_out, N*N*sizeof(float));
    if (err != hipSuccess){
       cout<<"Dev Memory not allocated"<<endl;
       exit(-1);
    }
     
    hipMemcpy(dev_in, host_in, N * N * sizeof(float), hipMemcpyHostToDevice);

    //create GPU timing events for timing the GPU
    hipEvent_t st2, et2;
    hipEventCreate(&st2);
    hipEventCreate(&et2);        
   
    int factor =N/64;
    dim3 grid(factor, factor);
    hipEventRecord(st2);
    kernel_call<<<grid, 128>>>(N, dev_in, dev_out);
    hipEventRecord(et2);
        
    //host waits until et2 has occured     
    hipEventSynchronize(et2);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, st2, et2);

    cout<<"Kernel time: "<<milliseconds<<"ms"<<endl;

    //copy data out
    hipMemcpy(host_out, dev_out, N * N * sizeof(float), hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i != N; ++i)
       for (int j = 0; j != N; ++j)
          correct &= (host_out[i*N+j] == host_in[j*N+i]);
    cout<<(correct ? "Yes" : "No")<<endl;	 

    correct = true;

   for (int tr = 0; tr < factor; tr++) {
      for (int tc = 0; tc < factor; tc++) {
         // Output tile at (tr, tc) is stored at:
         int out_tile_index = tr * factor + tc;
         // That tile came from input tile (tc, tr):
         int in_tile_index = tc * factor + tr;
         for (int a = 0; a < 64; a++) {
               for (int b = 0; b < 64; b++) {
                  // Index within the tile (row-major order)
                  int out_index = out_tile_index * 64*64 + a * 64 + b;
                  int in_index  = in_tile_index * 64*64 + b * 64 + a;
                  if (host_out[out_index] != host_in[in_index])
                     correct = false;
               }
         }
      }
   }

   cout << (correct ? "Yes" : "No") << endl;
   
    hipEventDestroy(st2);
    hipEventDestroy(et2);

    free(host_in);
    free(host_out);
    hipFree(dev_in);
    hipFree(dev_out);

  return 0;
}
